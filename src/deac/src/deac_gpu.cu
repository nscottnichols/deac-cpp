#include "hip/hip_runtime.h"
/**
 * @file deac_gpu.cu
 * @author Nathan Nichols
 * @date 04.19.2021
 *
 * @brief GPU kernels using CUDA.
 */

#include "deac_gpu.cuh"

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
// GPU KERNELS ---------------------------------------------------------------
// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------

__device__
uint64_t gpu_rol64(uint64_t x, uint64_t k) {
    return (x << k) | (x >> (64 - k));
}

__device__
uint64_t gpu_xoshiro256p_next(uint64_t * s) {
    uint64_t const result = s[0] + s[3];
    uint64_t const t = s[1] << 17;

    s[2] ^= s[0];
    s[3] ^= s[1];
    s[1] ^= s[2];
    s[0] ^= s[3];

    s[2] ^= t;
    s[3] = gpu_rol64(s[3], 45);

    return result;
}

__device__
void sub_group_reduce_add(volatile double* _c, size_t local_idx) {
    #if (SUB_GROUP_SIZE >= 64)
        _c[local_idx] += _c[local_idx + 64];
    #endif
    #if (SUB_GROUP_SIZE >= 32)
        _c[local_idx] += _c[local_idx + 32];
    #endif
    #if (SUB_GROUP_SIZE >= 16)
        _c[local_idx] += _c[local_idx + 16];
    #endif
    #if (SUB_GROUP_SIZE >= 8)
        _c[local_idx] += _c[local_idx + 8];
    #endif
    #if (SUB_GROUP_SIZE >= 4)
        _c[local_idx] += _c[local_idx + 4];
    #endif
    #if (SUB_GROUP_SIZE >= 2)
        _c[local_idx] += _c[local_idx + 2];
    #endif
    #if (SUB_GROUP_SIZE >= 1)
        _c[local_idx] += _c[local_idx + 1];
    #endif
}

__device__
void gpu_reduce_add(double* _c) {
    size_t local_idx = threadIdx.x;
    #if (GPU_BLOCK_SIZE >= 1024) && (SUB_GROUP_SIZE < 512)
        if (local_idx < 512) {
            _c[local_idx] += _c[local_idx + 512];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 512) && (SUB_GROUP_SIZE < 256)
        if (local_idx < 256) {
            _c[local_idx] += _c[local_idx + 256];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 256) && (SUB_GROUP_SIZE < 128)
        if (local_idx < 128) {
            _c[local_idx] += _c[local_idx + 128];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 128) && (SUB_GROUP_SIZE < 64)
        if (local_idx < 64) {
            _c[local_idx] += _c[local_idx + 64];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 64) && (SUB_GROUP_SIZE < 32)
        if (local_idx < 32) {
            _c[local_idx] += _c[local_idx + 32];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 32) && (SUB_GROUP_SIZE < 16)
        if (local_idx < 16) {
            _c[local_idx] += _c[local_idx + 16];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 16) && (SUB_GROUP_SIZE < 8)
        if (local_idx < 8) {
            _c[local_idx] += _c[local_idx + 8];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 8) && (SUB_GROUP_SIZE < 4)
        if (local_idx < 4) {
            _c[local_idx] += _c[local_idx + 4];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 4) && (SUB_GROUP_SIZE < 2)
        if (local_idx < 2) {
            _c[local_idx] += _c[local_idx + 2];
        }
        __syncthreads();
    #endif

    //Sub-group reduce
    if (local_idx < SUB_GROUP_SIZE) {
        sub_group_reduce_add(_c, local_idx);
    }
    __syncthreads();
}

__device__
void sub_group_reduce_min(volatile double* _c, size_t local_idx) {
    #if (SUB_GROUP_SIZE >= 64)
        _c[local_idx] = _c[local_idx + 64] < _c[local_idx] ? _c[local_idx + 64] : _c[local_idx];
    #endif
    #if (SUB_GROUP_SIZE >= 32)
        _c[local_idx] = _c[local_idx + 32] < _c[local_idx] ? _c[local_idx + 32] : _c[local_idx];
    #endif
    #if (SUB_GROUP_SIZE >= 16)
        _c[local_idx] = _c[local_idx + 16] < _c[local_idx] ? _c[local_idx + 16] : _c[local_idx];
    #endif
    #if (SUB_GROUP_SIZE >= 8)
        _c[local_idx] = _c[local_idx + 8] < _c[local_idx] ? _c[local_idx + 8] : _c[local_idx];
    #endif
    #if (SUB_GROUP_SIZE >= 4)
        _c[local_idx] = _c[local_idx + 4] < _c[local_idx] ? _c[local_idx + 4] : _c[local_idx];
    #endif
    #if (SUB_GROUP_SIZE >= 2)
        _c[local_idx] = _c[local_idx + 2] < _c[local_idx] ? _c[local_idx + 2] : _c[local_idx];
    #endif
    #if (SUB_GROUP_SIZE >= 1)
        _c[local_idx] = _c[local_idx + 1] < _c[local_idx] ? _c[local_idx + 1] : _c[local_idx];
    #endif
}

__device__
void gpu_reduce_min(double* _c) {
    size_t local_idx = threadIdx.x;
    #if (GPU_BLOCK_SIZE >= 1024) && (SUB_GROUP_SIZE < 512)
        if (local_idx < 512) {
            _c[local_idx] = _c[local_idx + 512] < _c[local_idx] ? _c[local_idx + 512] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 512) && (SUB_GROUP_SIZE < 256)
        if (local_idx < 256) {
            _c[local_idx] = _c[local_idx + 256] < _c[local_idx] ? _c[local_idx + 256] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 256) && (SUB_GROUP_SIZE < 128)
        if (local_idx < 128) {
            _c[local_idx] = _c[local_idx + 128] < _c[local_idx] ? _c[local_idx + 128] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 128) && (SUB_GROUP_SIZE < 64)
        if (local_idx < 64) {
            _c[local_idx] = _c[local_idx + 64] < _c[local_idx] ? _c[local_idx + 64] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 64) && (SUB_GROUP_SIZE < 32)
        if (local_idx < 32) {
            _c[local_idx] = _c[local_idx + 32] < _c[local_idx] ? _c[local_idx + 32] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 32) && (SUB_GROUP_SIZE < 16)
        if (local_idx < 16) {
            _c[local_idx] = _c[local_idx + 16] < _c[local_idx] ? _c[local_idx + 16] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 16) && (SUB_GROUP_SIZE < 8)
        if (local_idx < 8) {
            _c[local_idx] = _c[local_idx + 8] < _c[local_idx] ? _c[local_idx + 8] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 8) && (SUB_GROUP_SIZE < 4)
        if (local_idx < 4) {
            _c[local_idx] = _c[local_idx + 4] < _c[local_idx] ? _c[local_idx + 4] : _c[local_idx];
        }
        __syncthreads();
    #endif

    #if (GPU_BLOCK_SIZE >= 4) && (SUB_GROUP_SIZE < 2)
        if (local_idx < 2) {
            _c[local_idx] = _c[local_idx + 2] < _c[local_idx] ? _c[local_idx + 2] : _c[local_idx];
        }
        __syncthreads();
    #endif

    //Sub-group reduce
    if (local_idx < SUB_GROUP_SIZE) {
        sub_group_reduce_min(_c, local_idx);
    }
    __syncthreads();
}

__global__
void gpu_dot(double* __restrict__ C, double* __restrict__ B, double* __restrict__ A, size_t N) {
    // C = B*A where [B] = 1xN and [A] = Nx1
    // Shared Local Memory _c
    __shared__ double _c[GPU_BLOCK_SIZE];
    // Set shared local memory _c
    size_t local_idx = threadIdx.x;
    if (local_idx < N) {
        _c[local_idx] = A[local_idx]*B[local_idx];
    } else {
        _c[local_idx] = 0.0;
    }

    for (size_t i = 1; i < (N + GPU_BLOCK_SIZE - 1)/GPU_BLOCK_SIZE; i++) {
        size_t j = GPU_BLOCK_SIZE*i + local_idx;
        if (j < N) {
            _c[local_idx] += A[j]*B[j];
        }
    }
    __syncthreads();

    // Reduce _c (using shared local memory)
    gpu_reduce_add(_c);

    //Set C
    if (local_idx == 0) {
         C[0] += _c[0]; //FIXME should do C[0] = _c[0] + scale_factor*C[0] here probably
    }
}

__global__ void gpu_matmul_simple(int m, int n, int k, double alpha, double* __restrict__ A, int lda, double* __restrict__ B, int ldb, double beta, double* __restrict__ C, int ldc) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        double sum = 0.0;
        for (int e = 0; e < k; e++) {
            sum += A[row + e * lda] * B[e + col * ldb];
        }
        C[row + col * ldc] = alpha * sum + beta * C[row + col * ldc];
    }
}

__global__ void gpu_matmul(int m, int n, int k, double alpha, double* __restrict__ A, int lda, double* __restrict__ B, int ldb, double beta, double* __restrict__ C, int ldc) {
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    // Identify the row and column of the C element to work on
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    double Cvalue = 0.0;

    // Loop over the A and B tiles required to compute the C element
    for (int t = 0; t < (k-1)/TILE_WIDTH + 1; ++t) {
        __shared__ double As[TILE_WIDTH][TILE_WIDTH];
        __shared__ double Bs[TILE_WIDTH][TILE_WIDTH];

        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (row < m && t*TILE_WIDTH+tx < k)
            As[ty][tx] = A[row + lda * (t*TILE_WIDTH+tx)];
        else
            As[ty][tx] = 0.0;

        if (t*TILE_WIDTH+ty < k && col < n)
            Bs[ty][tx] = B[(t*TILE_WIDTH+ty) + ldb * col];
        else
            Bs[ty][tx] = 0.0;

        __syncthreads(); // Make sure the matrices are loaded before starting the computation

        // Multiply the two matrices together; each thread computes one element of the block sub-matrix
        for (int e = 0; e < TILE_WIDTH; ++e) {
            Cvalue += As[ty][e] * Bs[e][tx];
        }

        __syncthreads(); // Make sure that all threads are done computing before loading the next set of tiles
    }

    if (row < m && col < n)
        C[row + ldc * col] = alpha * Cvalue + beta * C[row + ldc * col];
}

__global__ void gpu_deac_gemv_simple(int m, int n, double alpha, double* __restrict__ A, int lda, double* __restrict__ x, int incx, double beta, double* __restrict__ y, int incy) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m) {
        double sum = 0.0;
        for (int j = 0; j < n; j++) {
            sum += A[row + j*lda] * x[j*incx];
        }
        y[row*incy] = alpha * sum + beta * y[row*incy];
    }
}

__global__ void gpu_deac_gemv_atomic(int m, int n, double alpha, double* __restrict__ A, int lda, double* __restrict__ x, int incx, double beta, double* __restrict__ y, int incy) {
    __shared__ double shared_x[TILE_WIDTH];
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < n) {
        shared_x[threadIdx.x] = x[col * incx];
    }
    __syncthreads();

    if (col < n) {
        for (int i = 0; i < m; i++) {
            double Aval = A[i + col * lda];
            atomicAdd(&y[i * incy], alpha * Aval * shared_x[threadIdx.x]);
        }
    }
}

__global__ void gpu_deac_gemv(int m, int n, double alpha, double* __restrict__ A, int lda, double* __restrict__ x, int incx, double beta, double* __restrict__ y, int incy) {
    __shared__ double As[TILE_WIDTH][TILE_WIDTH];
    int tx = threadIdx.x;
    int by = blockIdx.y, ty = threadIdx.y;
    int row = by * blockDim.y + ty;

    double sum = 0.0;
    if (row < m) {
        for (int i = 0; i < (n + TILE_WIDTH - 1) / TILE_WIDTH; ++i) {
            if (i*TILE_WIDTH + tx < n && row < m) {
                As[ty][tx] = A[row + (i*TILE_WIDTH + tx) * lda];
            } else {
                As[ty][tx] = 0.0;
            }
            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; ++k) {
                if (i*TILE_WIDTH + k < n) {
                    sum += As[ty][k] * x[(i*TILE_WIDTH + k)*incx];
                }
            }
            __syncthreads();
        }
        if (beta == 0.0) {
            y[row * incy] = alpha * sum;
        } else {
            y[row * incy] = alpha * sum + beta * y[row * incy];
        }
    }
}

__global__
void gpu_get_minimum(double* __restrict__ minimum, double* __restrict__ array, size_t N) {
    // finds minimum of array with length N
    // Shared Local Memory _c
    __shared__ double _c[GPU_BLOCK_SIZE];
    // Set shared local memory _c
    size_t local_idx = threadIdx.x;
    if (local_idx < N) {
        _c[local_idx] = array[local_idx];
    } else {
        _c[local_idx] = array[0];
    }

    for (size_t i = 1; i < (N + GPU_BLOCK_SIZE - 1)/GPU_BLOCK_SIZE; i++) {
        size_t j = GPU_BLOCK_SIZE*i + local_idx;
        if (j < N) {
            _c[local_idx] = array[j] < _c[local_idx] ? array[j] : _c[local_idx];
        }
    }
    __syncthreads();

    // Reduce _c (using shared local memory)
    gpu_reduce_min(_c);

    //Set minimum
    if (local_idx == 0) {
         minimum[0] += _c[0];
    }
}

__global__
void gpu_normalize_population(double* __restrict__ population, double* __restrict__ normalization, double zeroth_moment, size_t population_size, size_t genome_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size*genome_size) {
        population[global_idx] *= zeroth_moment/normalization[global_idx/genome_size];
    }
}

__global__
void gpu_set_fitness(double* __restrict__ fitness, double* __restrict__ isf, double* __restrict__ isf_model, double* __restrict__ isf_error, size_t number_of_timeslices) {
    __shared__ double _f[GPU_BLOCK_SIZE];
    // Set shared local memory _f
    size_t local_idx = threadIdx.x;
    if (local_idx < number_of_timeslices) {
        double __f = (isf[local_idx] - isf_model[local_idx])/isf_error[local_idx];
        _f[local_idx] = __f*__f;
    } else {
        _f[local_idx] = 0.0;
    }

    for (size_t i = 1; i < (number_of_timeslices + GPU_BLOCK_SIZE - 1)/GPU_BLOCK_SIZE; i++) {
        size_t j = GPU_BLOCK_SIZE*i + local_idx;
        if (j < number_of_timeslices) {
            double __f = (isf[j] - isf_model[j])/isf_error[j];
            _f[local_idx] += __f*__f;
        }
    }
    __syncthreads();

    // Reduce _f (using shared local memory)
    gpu_reduce_add(_f);

    //Set fitness
    if (local_idx == 0) {
         fitness[0] += _f[0]/number_of_timeslices;
    }
}

__global__
void gpu_set_fitness_moments_reduced_chi_squared(double* __restrict__ fitness, double* __restrict__ moments, double moment, double moment_error, size_t population_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        double __f = (moment - moments[global_idx])/moment_error;
        fitness[global_idx] += __f*__f;
    }
}

__global__
void gpu_set_fitness_moments_chi_squared(double* __restrict__ fitness, double* __restrict__ moments, double moment, size_t population_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        double __f = moment - moments[global_idx];
        fitness[global_idx] += __f*__f;
    }
}

__global__
void gpu_set_fitness_mean(double* __restrict__ fitness_mean, double* __restrict__ fitness, size_t population_size) {
    __shared__ double _fm[GPU_BLOCK_SIZE];
    // Set shared local memory _fm
    size_t local_idx = threadIdx.x;
    if (local_idx < population_size) {
        _fm[local_idx] = fitness[local_idx];
    } else {
        _fm[local_idx] = 0.0;
    }

    for (size_t i = 1; i < (population_size + GPU_BLOCK_SIZE - 1)/GPU_BLOCK_SIZE; i++) {
        size_t j = GPU_BLOCK_SIZE*i + local_idx;
        if (j < population_size) {
            _fm[local_idx] += fitness[j];
        }
    }
    __syncthreads();
    
    // Reduce _fm (using shared local memory)
    gpu_reduce_add(_fm);

    //Set fitness_mean
    if (local_idx == 0) {
         fitness_mean[0] += _fm[0]/population_size;
    }
}

__global__
void gpu_set_fitness_squared_mean(double* __restrict__ fitness_squared_mean, double* __restrict__ fitness, size_t population_size) {
    __shared__ double _fsm[GPU_BLOCK_SIZE];
    // Set shared local memory _fsm
    size_t local_idx = threadIdx.x;
    if (local_idx < population_size) {
        _fsm[local_idx] = fitness[local_idx]*fitness[local_idx];
    } else {
        _fsm[local_idx] = 0.0;
    }

    for (size_t i = 1; i < (population_size + GPU_BLOCK_SIZE - 1)/GPU_BLOCK_SIZE; i++) {
        size_t j = GPU_BLOCK_SIZE*i + local_idx;
        if (j < population_size) {
            _fsm[local_idx] += fitness[j]*fitness[j];
        }
    }
    __syncthreads();
    
    // Reduce _fsm (using shared local memory)
    gpu_reduce_add(_fsm);

    //Set fitness_squared_mean
    if (local_idx == 0) {
         fitness_squared_mean[0] += _fsm[0]/population_size;
    }
}

__global__
void gpu_set_population_new(double* __restrict__ population_new, double* __restrict__ population_old, size_t* __restrict__ mutant_indices, double* __restrict__ differential_weights_new, bool* __restrict__ mutate_indices, size_t population_size, size_t genome_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size*genome_size) {
        size_t _i = global_idx/genome_size;
        size_t _j = global_idx - _i*genome_size;
        double F = differential_weights_new[_i];
        size_t mutant_index1 = mutant_indices[3*_i];
        size_t mutant_index2 = mutant_indices[3*_i + 1];
        size_t mutant_index3 = mutant_indices[3*_i + 2];
        bool mutate = mutate_indices[global_idx];
        if (mutate) {
            #ifdef ALLOW_NEGATIVE_SPECTRAL_WEIGHT
                population_new[global_idx] = population_old[mutant_index1*genome_size + _j] + F*(population_old[mutant_index2*genome_size + _j] - population_old[mutant_index3*genome_size + _j]);
            #else
                population_new[global_idx] = fabs(population_old[mutant_index1*genome_size + _j] + F*(population_old[mutant_index2*genome_size + _j] - population_old[mutant_index3*genome_size + _j]));
            #endif
        } else {
            population_new[global_idx] = population_old[global_idx];
        }
    }
}


__global__
void gpu_match_population_zero(double* __restrict__ population_negative_frequency, double* __restrict__ population_positive_frequency, size_t population_size, size_t genome_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        population_negative_frequency[global_idx*genome_size] = population_positive_frequency[global_idx*genome_size];
    }
}

__global__
void gpu_set_rejection_indices(bool* __restrict__ rejection_indices, double* __restrict__ fitness_new, double* __restrict__ fitness_old, size_t population_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        bool accept = fitness_new[global_idx] <= fitness_old[global_idx];
        rejection_indices[global_idx] = accept;
        if (accept) {
            fitness_old[global_idx] = fitness_new[global_idx];
        }
    }
}

__global__
void gpu_swap_control_parameters(double* __restrict__ control_parameter_old, double* __restrict__ control_parameter_new, bool* __restrict__ rejection_indices, size_t population_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        if (rejection_indices[global_idx]) {
            control_parameter_old[global_idx] = control_parameter_new[global_idx];
        }
    }
}

__global__
void gpu_swap_populations(double* __restrict__ population_old, double* __restrict__ population_new, bool* __restrict__ rejection_indices, size_t population_size, size_t genome_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size*genome_size) {
        size_t _i = global_idx/genome_size;
        if (rejection_indices[_i]) {
            population_old[global_idx] = population_new[global_idx];
        }
    }
}

__global__
void gpu_set_crossover_probabilities_new(uint64_t* __restrict__ rng_state, double* __restrict__ crossover_probabilities_new, double* __restrict__ crossover_probabilities_old, double self_adapting_crossover_probability, size_t population_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        if ((gpu_xoshiro256p_next(rng_state + 4*global_idx) >> 11) * 0x1.0p-53 < self_adapting_crossover_probability) {
            crossover_probabilities_new[global_idx] = (gpu_xoshiro256p_next(rng_state + 4*global_idx) >> 11) * 0x1.0p-53;
        } else {
            crossover_probabilities_new[global_idx] = crossover_probabilities_old[global_idx];
        }
    }
}

__global__
void gpu_set_differential_weights_new(uint64_t* __restrict__ rng_state, double* __restrict__ differential_weights_new, double* __restrict__ differential_weights_old, double self_adapting_differential_weight_probability, size_t population_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        if ((gpu_xoshiro256p_next(rng_state + 4*global_idx) >> 11) * 0x1.0p-53 < self_adapting_differential_weight_probability) {
            differential_weights_new[global_idx] = 2.0*((gpu_xoshiro256p_next(rng_state + 4*global_idx) >> 11) * 0x1.0p-53);
        } else {
            differential_weights_new[global_idx] = differential_weights_old[global_idx];
        }
    }
}

__device__
void gpu_set_mutant_indices(uint64_t* __restrict__ rng_state, size_t* __restrict__ mutant_indices, size_t mutant_index0, size_t length) {
    mutant_indices[0] = mutant_index0;
    mutant_indices[1] = mutant_index0;
    mutant_indices[2] = mutant_index0;
    while (mutant_indices[0] == mutant_index0) {
        mutant_indices[0] = gpu_xoshiro256p_next(rng_state) % length;
    }

    while ((mutant_indices[1] == mutant_index0) || (mutant_indices[1] == mutant_indices[0])) {
        mutant_indices[1] = gpu_xoshiro256p_next(rng_state) % length;
    }

    while ((mutant_indices[2] == mutant_index0) || (mutant_indices[2] == mutant_indices[0])
            || (mutant_indices[2] == mutant_indices[1])) {
        mutant_indices[2] = gpu_xoshiro256p_next(rng_state) % length;
    }
}

__global__
void gpu_set_mutant_indices(uint64_t* __restrict__ rng_state, size_t* __restrict__ mutant_indices, size_t population_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size) {
        gpu_set_mutant_indices(rng_state + 4*global_idx, mutant_indices + 3*global_idx, global_idx, population_size);
    }
}

__global__
void gpu_set_mutate_indices(uint64_t* __restrict__ rng_state, bool* __restrict__ mutate_indices, double* __restrict__ crossover_probabilities, size_t population_size, size_t genome_size) {
    size_t global_idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (global_idx < population_size*genome_size) {
        size_t _i = global_idx/genome_size;
        mutate_indices[global_idx] = (gpu_xoshiro256p_next(rng_state + 4*global_idx) >> 11) * 0x1.0p-53 < crossover_probabilities[_i];
    }
}

// Kernel Launcher
void gpu_dot(hipStream_t s, double* __restrict__ C, double* __restrict__ B, double* __restrict__ A, size_t N) {
    gpu_dot<<<dim3(1), dim3(GPU_BLOCK_SIZE), 0, s>>>(C, B, A, N);
}

void gpu_matmul(hipStream_t s, int m, int n, int k, double alpha, double* __restrict__ A, double* __restrict__ B, double beta, double* __restrict__ C) {
    gpu_matmul<<<dim3((n + TILE_WIDTH - 1) / TILE_WIDTH, (m + TILE_WIDTH - 1) / TILE_WIDTH), dim3(TILE_WIDTH, TILE_WIDTH), 0, s>>>(m, n, k, alpha, A, m, B, k, beta, C, m);
}

void gpu_deac_gemv(hipStream_t s, int m, int n, double alpha, double* __restrict__ A, double* __restrict__ x, double beta, double* __restrict__ y) {
    //gpu_deac_gemv_simple<<<dim3((m + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE), dim3(GPU_BLOCK_SIZE), 0, s>>>(m, n, alpha, A, m, x, 1, beta, y, 1);
    //gpu_deac_gemv_atomic<<<dim3((n + TILE_WIDTH - 1) / TILE_WIDTH), dim3(TILE_WIDTH), 0, s>>>(m, n, alpha, A, m, x, 1, beta, y, 1);
    gpu_deac_gemv<<<dim3((n + TILE_WIDTH - 1) / TILE_WIDTH, (n + TILE_WIDTH - 1) / TILE_WIDTH), dim3(TILE_WIDTH, TILE_WIDTH), 0, s>>>(m, n, alpha, A, m, x, 1, beta, y, 1);
}

void gpu_get_minimum(hipStream_t s, double* __restrict__ minimum, double* __restrict__ array, size_t N) {
    gpu_get_minimum<<<dim3(1), dim3(GPU_BLOCK_SIZE), 0, s>>>(minimum, array, N);
}

void gpu_normalize_population(hipStream_t s, size_t grid_size, double* __restrict__ population, double* __restrict__ normalization, double zeroth_moment, size_t population_size, size_t genome_size) {
    gpu_normalize_population<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(population, normalization, zeroth_moment, population_size, genome_size);
}

void gpu_set_fitness(hipStream_t s, double* __restrict__ fitness, double* __restrict__ isf, double* __restrict__ isf_model, double* __restrict__ isf_error, size_t number_of_timeslices) {
    gpu_set_fitness<<<dim3(1), dim3(GPU_BLOCK_SIZE), 0, s>>>(fitness, isf, isf_model, isf_error, number_of_timeslices);
}

void gpu_set_fitness_moments_reduced_chi_squared(hipStream_t s, size_t grid_size, double* __restrict__ fitness, double* __restrict__ moments, double moment, double moment_error, size_t population_size) {
    gpu_set_fitness_moments_reduced_chi_squared<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(fitness, moments, moment, moment_error, population_size);
}

void gpu_set_fitness_moments_chi_squared(hipStream_t s, size_t grid_size, double* __restrict__ fitness, double* __restrict__ moments, double moment, size_t population_size) {
    gpu_set_fitness_moments_chi_squared<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(fitness, moments, moment, population_size);
}

void gpu_set_fitness_mean(hipStream_t s, double* __restrict__ fitness_mean, double* __restrict__ fitness, size_t population_size) {
    gpu_set_fitness_mean<<<dim3(1), dim3(GPU_BLOCK_SIZE), 0, s>>>(fitness_mean, fitness, population_size);
}

void gpu_set_fitness_squared_mean(hipStream_t s, double* __restrict__ fitness_squared_mean, double* __restrict__ fitness, size_t population_size) {
    gpu_set_fitness_squared_mean<<<dim3(1), dim3(GPU_BLOCK_SIZE), 0, s>>>(fitness_squared_mean, fitness, population_size);
}

void gpu_set_population_new(hipStream_t s, size_t grid_size, double* __restrict__ population_new, double* __restrict__ population_old, size_t* __restrict__ mutant_indices, double* __restrict__ differential_weights_new, bool* __restrict__ mutate_indices, size_t population_size, size_t genome_size) {
    gpu_set_population_new<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(population_new, population_old, mutant_indices, differential_weights_new, mutate_indices, population_size, genome_size);
}

void gpu_match_population_zero(hipStream_t s, size_t grid_size, double* __restrict__ population_negative_frequency, double* __restrict__ population_positive_frequency, size_t population_size, size_t genome_size) {
    gpu_match_population_zero<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(population_negative_frequency, population_positive_frequency, population_size, genome_size);
}

void gpu_set_rejection_indices(hipStream_t s, size_t grid_size, bool* __restrict__ rejection_indices, double* __restrict__ fitness_new, double* __restrict__ fitness_old, size_t population_size) {
    gpu_set_rejection_indices<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(rejection_indices, fitness_new, fitness_old, population_size);
}

void gpu_swap_control_parameters(hipStream_t s, size_t grid_size, double* __restrict__ control_parameter_old, double* __restrict__ control_parameter_new, bool* __restrict__ rejection_indices, size_t population_size) {
    gpu_swap_control_parameters<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(control_parameter_old, control_parameter_new, rejection_indices, population_size);
}

void gpu_swap_populations(hipStream_t s, size_t grid_size, double* __restrict__ population_old, double* __restrict__ population_new, bool* __restrict__ rejection_indices, size_t population_size, size_t genome_size) {
    gpu_swap_populations<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(population_old, population_new, rejection_indices, population_size, genome_size);
}

void gpu_set_crossover_probabilities_new(hipStream_t s, size_t grid_size, uint64_t* __restrict__ rng_state, double* __restrict__ crossover_probabilities_new, double* __restrict__ crossover_probabilities_old, double self_adapting_crossover_probability, size_t population_size) {
    gpu_set_crossover_probabilities_new<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(rng_state, crossover_probabilities_new, crossover_probabilities_old, self_adapting_crossover_probability, population_size);
}

void gpu_set_differential_weights_new(hipStream_t s, size_t grid_size, uint64_t* __restrict__ rng_state, double* __restrict__ differential_weights_new, double* __restrict__ differential_weights_old, double self_adapting_differential_weight_probability, size_t population_size) {
    gpu_set_differential_weights_new<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(rng_state, differential_weights_new, differential_weights_old, self_adapting_differential_weight_probability, population_size);
}

void gpu_set_mutant_indices(hipStream_t s, size_t grid_size, uint64_t* __restrict__ rng_state, size_t* __restrict__ mutant_indices, size_t population_size) {
    gpu_set_mutant_indices<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(rng_state, mutant_indices, population_size);
}

void gpu_set_mutate_indices(hipStream_t s, size_t grid_size, uint64_t* __restrict__ rng_state, bool* __restrict__ mutate_indices, double* __restrict__ crossover_probabilities, size_t population_size, size_t genome_size) {
    gpu_set_mutate_indices<<<dim3(grid_size), dim3(GPU_BLOCK_SIZE), 0, s>>>(rng_state, mutate_indices, crossover_probabilities, population_size, genome_size);
}

#ifdef USE_BLAS
    void gpu_blas_gemv(hipblasHandle_t handle, int m, int n, double alpha, double* A, double* B, double beta, double* C) {
        GPU_BLAS_ASSERT(hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha, A, m, B, 1, &beta, C, 1));
    }

    void gpu_blas_gemm(hipblasHandle_t handle, int m, int n, int k, double alpha, double* A, double* B, double beta, double* C) {
        GPU_BLAS_ASSERT(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, m, B, k, &beta, C, m));
    }
#endif
