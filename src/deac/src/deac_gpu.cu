#include "hip/hip_runtime.h"
/**
 * @file deac_gpu.hip.hpp
 * @author Nathan Nichols
 * @date 04.19.2021
 *
 * @brief GPU kernels using CUDA.
 */

#include "deac_gpu.cuh"
#ifdef DEAC_DEBUG
    #include <stdio.h>
#endif
//

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
// GPU KERNELS ---------------------------------------------------------------
// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------

#ifdef DEAC_DEBUG
    __global__
    void gpu_check_array(double * _array, int length) {
        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if (i < length) {
            printf("_array[%d]: %e\n", i, _array[i]);
        }
    }
#endif

// GPU Kernel for reduction using warp (uses appropriate warp for NVIDIA vs AMD devices i. e. "portable wave aware code")
__device__ void warp_reduce(volatile double *sdata, unsigned int thread_idx) {
    if (warpSize == 64) { if (GPU_BLOCK_SIZE >= 128) sdata[thread_idx] += sdata[thread_idx + 64]; }
    if (GPU_BLOCK_SIZE >= 64) sdata[thread_idx] += sdata[thread_idx + 32];
    if (GPU_BLOCK_SIZE >= 32) sdata[thread_idx] += sdata[thread_idx + 16];
    if (GPU_BLOCK_SIZE >= 16) sdata[thread_idx] += sdata[thread_idx + 8];
    if (GPU_BLOCK_SIZE >= 8) sdata[thread_idx] += sdata[thread_idx + 4];
    if (GPU_BLOCK_SIZE >= 4) sdata[thread_idx] += sdata[thread_idx + 2];
    if (GPU_BLOCK_SIZE >= 2) sdata[thread_idx] += sdata[thread_idx + 1];
}

__device__ void warp_reduce_min(volatile double *sdata, unsigned int thread_idx) {
    if (warpSize == 64) { if (GPU_BLOCK_SIZE >= 128) sdata[thread_idx] = sdata[thread_idx + 64] < sdata[thread_idx] ? sdata[thread_idx + 64] : sdata[thread_idx]; }
    if (GPU_BLOCK_SIZE >= 64) sdata[thread_idx] = sdata[thread_idx + 32] < sdata[thread_idx] ? sdata[thread_idx + 32] : sdata[thread_idx];
    if (GPU_BLOCK_SIZE >= 32) sdata[thread_idx] = sdata[thread_idx + 16] < sdata[thread_idx] ? sdata[thread_idx + 16] : sdata[thread_idx];
    if (GPU_BLOCK_SIZE >= 16) sdata[thread_idx] = sdata[thread_idx + 8] < sdata[thread_idx] ? sdata[thread_idx + 8] : sdata[thread_idx];
    if (GPU_BLOCK_SIZE >= 8) sdata[thread_idx] = sdata[thread_idx + 4] < sdata[thread_idx] ? sdata[thread_idx + 4] : sdata[thread_idx];
    if (GPU_BLOCK_SIZE >= 4) sdata[thread_idx] = sdata[thread_idx + 2] < sdata[thread_idx] ? sdata[thread_idx + 2] : sdata[thread_idx];
    if (GPU_BLOCK_SIZE >= 2) sdata[thread_idx] = sdata[thread_idx + 1] < sdata[thread_idx] ? sdata[thread_idx + 1] : sdata[thread_idx];
}

__global__
void gpu_matrix_multiply_MxN_by_Nx1(double * C, double * A, double * B, int N, int idx) {
    __shared__ double _c[GPU_BLOCK_SIZE];
    int _j = blockDim.x * blockIdx.x + threadIdx.x;
    if (_j < N) {
        _c[threadIdx.x] = A[idx*N + _j]*B[_j];
    } else {
        _c[threadIdx.x] = 0.0;
    }
    __syncthreads();

    // NEED TO REDUCE _c ON SHARED MEMORY AND ADD TO GLOBAL isf
    if (GPU_BLOCK_SIZE >= 1024) {
        if (threadIdx.x < 512) {
            _c[threadIdx.x] += _c[threadIdx.x + 512];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 512) {
        if (threadIdx.x < 256) {
            _c[threadIdx.x] += _c[threadIdx.x + 256];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 256) {
        if (threadIdx.x < 128) {
            _c[threadIdx.x] += _c[threadIdx.x + 128];
        }
        __syncthreads();
    } 

    if (warpSize == 32) {
        if (GPU_BLOCK_SIZE >= 128) {
            if (threadIdx.x < 64) {
                _c[threadIdx.x] += _c[threadIdx.x + 64];
            }
            __syncthreads();
        } 
    }

    if (threadIdx.x < warpSize) {
        warp_reduce(_c, threadIdx.x);
    }

    if (threadIdx.x == 0) {
        //NOTE: May see some performance gain here if temporarily store results
        // to some global device variable and then launch a separate kernel to
        // again reduce those results i.e.
        // tmp_c[blockIdx.x] = _c[0];
        // ^-- reduce on this, but this code may get too bloated
        atomicAdd(&C[idx], _c[0]);
    }
}

__global__
void gpu_matrix_multiply_LxM_by_MxN(double * C, double * A, double * B, int L, int M, int idx) {
    __shared__ double _c[GPU_BLOCK_SIZE];
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if (k < M) {
        int _i = idx/L;
        int _j = idx - _i*L;
        _c[threadIdx.x] = A[_j*M + k]*B[_i*M + k];
    } else {
        _c[threadIdx.x] = 0.0;
    }
    __syncthreads();

    // NEED TO REDUCE _c ON SHARED MEMORY AND ADD TO GLOBAL isf
    if (GPU_BLOCK_SIZE >= 1024) {
        if (threadIdx.x < 512) {
            _c[threadIdx.x] += _c[threadIdx.x + 512];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 512) {
        if (threadIdx.x < 256) {
            _c[threadIdx.x] += _c[threadIdx.x + 256];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 256) {
        if (threadIdx.x < 128) {
            _c[threadIdx.x] += _c[threadIdx.x + 128];
        }
        __syncthreads();
    } 

    if (warpSize == 32) {
        if (GPU_BLOCK_SIZE >= 128) {
            if (threadIdx.x < 64) {
                _c[threadIdx.x] += _c[threadIdx.x + 64];
            }
            __syncthreads();
        } 
    }

    if (threadIdx.x < warpSize) {
        warp_reduce(_c, threadIdx.x);
    }

    if (threadIdx.x == 0) {
        //NOTE: May see some performance gain here if temporarily store results
        // to some global device variable and then launch a separate kernel to
        // again reduce those results i.e.
        // tmp_c[blockIdx.x] = _c[0];
        // ^-- reduce on this, but this code may get too bloated
        atomicAdd(&C[idx], _c[0]);
    }
}

__global__
void gpu_normalize_population(double * population, double * normalization, double zeroth_moment, int population_size, int genome_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size*genome_size) {
        double _norm = normalization[i/genome_size];
        population[i] *= zeroth_moment/_norm;
    }
}

__global__
void gpu_set_fitness(double * fitness, double * isf, double * isf_model, double * isf_error, int number_of_timeslices, int idx) {
    __shared__ double _f[GPU_BLOCK_SIZE];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < number_of_timeslices) {
        _f[threadIdx.x] = pow((isf[i] - isf_model[idx*number_of_timeslices + i])/isf_error[i],2);
    } else {
        _f[threadIdx.x] = 0.0;
    }
    __syncthreads();

    // NEED TO REDUCE _f ON SHARED MEMORY AND ADD TO GLOBAL isf
    if (GPU_BLOCK_SIZE >= 1024) {
        if (threadIdx.x < 512) {
            _f[threadIdx.x] += _f[threadIdx.x + 512];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 512) {
        if (threadIdx.x < 256) {
            _f[threadIdx.x] += _f[threadIdx.x + 256];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 256) {
        if (threadIdx.x < 128) {
            _f[threadIdx.x] += _f[threadIdx.x + 128];
        }
        __syncthreads();
    } 

    if (warpSize == 32) {
        if (GPU_BLOCK_SIZE >= 128) {
            if (threadIdx.x < 64) {
                _f[threadIdx.x] += _f[threadIdx.x + 64];
            }
            __syncthreads();
        } 
    }

    if (threadIdx.x < warpSize) {
        warp_reduce(_f, threadIdx.x);
    }

    if (threadIdx.x == 0) {
        //NOTE: May see some performance gain here if temporarily store results
        // to some global device variable and then launch a separate kernel to
        // again reduce those results i.e.
        // tmp_f[blockIdx.x] = _f[0];
        // ^-- reduce on this, but this code may get too bloated
        atomicAdd(&fitness[idx], _f[0]/number_of_timeslices);
    }
}

__global__
void gpu_set_fitness_moments_reduced_chi_squared(double * fitness, double * moments, double moment, double moment_error, int population_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size) {
        fitness[i] += pow((moment - moments[i])/moment_error,2);
    } 
}

__global__
void gpu_set_fitness_moments_chi_squared(double * fitness, double * moments, double moment, int population_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size) {
        fitness[i] += pow((moment - moments[i]),2)/moment;
    } 
}

__global__
void gpu_get_minimum_fitness(double * fitness, double * minimum_fitness, int population_size) {
    __shared__ double s_minimum[GPU_BLOCK_SIZE];
    if (threadIdx.x < population_size) {
        s_minimum[threadIdx.x] = fitness[threadIdx.x];
    } else {
        s_minimum[threadIdx.x] = fitness[0];
    }

    for (int i=0; i<population_size/GPU_BLOCK_SIZE; i++) {
        int j = GPU_BLOCK_SIZE*i + threadIdx.x;
        if (j < population_size) {
            s_minimum[threadIdx.x] = fitness[j] < s_minimum[threadIdx.x] ? fitness[j] : s_minimum[threadIdx.x];
        }
    }

    __syncthreads();

    // NEED TO REDUCE _f ON SHARED MEMORY AND ADD TO GLOBAL isf
    if (GPU_BLOCK_SIZE >= 1024) {
        if (threadIdx.x < 512) {
            s_minimum[threadIdx.x] = s_minimum[threadIdx.x + 512] < s_minimum[threadIdx.x] ? s_minimum[threadIdx.x + 512] : s_minimum[threadIdx.x];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 512) {
        if (threadIdx.x < 256) {
            s_minimum[threadIdx.x] = s_minimum[threadIdx.x + 256] < s_minimum[threadIdx.x] ? s_minimum[threadIdx.x + 256] : s_minimum[threadIdx.x];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 256) {
        if (threadIdx.x < 128) {
            s_minimum[threadIdx.x] = s_minimum[threadIdx.x + 128] < s_minimum[threadIdx.x] ? s_minimum[threadIdx.x + 128] : s_minimum[threadIdx.x];
        }
        __syncthreads();
    } 

    if (warpSize == 32) {
        if (GPU_BLOCK_SIZE >= 128) {
            if (threadIdx.x < 64) {
                s_minimum[threadIdx.x] = s_minimum[threadIdx.x + 64] < s_minimum[threadIdx.x] ? s_minimum[threadIdx.x + 64] : s_minimum[threadIdx.x];
            }
            __syncthreads();
        } 
    }

    if (threadIdx.x < warpSize) {
        warp_reduce_min(s_minimum, threadIdx.x);
    }

    if (threadIdx.x == 0) {
        *minimum_fitness = s_minimum[0];
    }

}

__global__
void gpu_set_fitness_mean(double * fitness_mean, double * fitness, int population_size, int idx) {
    __shared__ double _f[GPU_BLOCK_SIZE];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size) {
        _f[threadIdx.x] = fitness[i];
    } else {
        _f[threadIdx.x] = 0.0;
    }
    __syncthreads();

    // NEED TO REDUCE _f ON SHARED MEMORY AND ADD TO GLOBAL isf
    if (GPU_BLOCK_SIZE >= 1024) {
        if (threadIdx.x < 512) {
            _f[threadIdx.x] += _f[threadIdx.x + 512];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 512) {
        if (threadIdx.x < 256) {
            _f[threadIdx.x] += _f[threadIdx.x + 256];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 256) {
        if (threadIdx.x < 128) {
            _f[threadIdx.x] += _f[threadIdx.x + 128];
        }
        __syncthreads();
    } 

    if (warpSize == 32) {
        if (GPU_BLOCK_SIZE >= 128) {
            if (threadIdx.x < 64) {
                _f[threadIdx.x] += _f[threadIdx.x + 64];
            }
            __syncthreads();
        } 
    }

    if (threadIdx.x < warpSize) {
        warp_reduce(_f, threadIdx.x);
    }

    if (threadIdx.x == 0) {
        //NOTE: May see some performance gain here if temporarily store results
        // to some global device variable and then launch a separate kernel to
        // again reduce those results i.e.
        // tmp_f[blockIdx.x] = _f[0];
        // ^-- reduce on this, but this code may get too bloated
        atomicAdd(&fitness_mean[idx], _f[0]/population_size);
    }
}

__global__
void gpu_set_fitness_standard_deviation(double * fitness_standard_deviation, double * fitness_mean, double * fitness, int population_size, int idx) {
    __shared__ double _f[GPU_BLOCK_SIZE];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size) {
        _f[threadIdx.x] = pow(fitness[i] - fitness_mean[idx],2);
    } else {
        _f[threadIdx.x] = 0.0;
    }
    __syncthreads();

    // NEED TO REDUCE _f ON SHARED MEMORY AND ADD TO GLOBAL isf
    if (GPU_BLOCK_SIZE >= 1024) {
        if (threadIdx.x < 512) {
            _f[threadIdx.x] += _f[threadIdx.x + 512];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 512) {
        if (threadIdx.x < 256) {
            _f[threadIdx.x] += _f[threadIdx.x + 256];
        }
        __syncthreads();
    } 

    if (GPU_BLOCK_SIZE >= 256) {
        if (threadIdx.x < 128) {
            _f[threadIdx.x] += _f[threadIdx.x + 128];
        }
        __syncthreads();
    } 

    if (warpSize == 32) {
        if (GPU_BLOCK_SIZE >= 128) {
            if (threadIdx.x < 64) {
                _f[threadIdx.x] += _f[threadIdx.x + 64];
            }
            __syncthreads();
        } 
    }

    if (threadIdx.x < warpSize) {
        warp_reduce(_f, threadIdx.x);
    }

    if (threadIdx.x == 0) {
        //NOTE: May see some performance gain here if temporarily store results
        // to some global device variable and then launch a separate kernel to
        // again reduce those results i.e.
        // tmp_f[blockIdx.x] = _f[0];
        // ^-- reduce on this, but this code may get too bloated
        atomicAdd(&fitness_mean[idx], _f[0]/population_size);
    }
}

__global__
void gpu_set_fitness_standard_deviation_sqrt(double * fitness_standard_deviation, int max_generations) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < max_generations) {
        fitness_standard_deviation[i] = sqrt(fitness_standard_deviation[i]);
    }
}

__global__
void gpu_set_population_new(double * population_new, double * population_old, int * mutant_indices, double * differential_weights_new, bool * mutate_indices, int population_size, int genome_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size*genome_size) {
        int _i = i/genome_size;
        int _j = i - _i*genome_size;
        double F = differential_weights_new[_i];
        int mutant_index1 = mutant_indices[3*_i];
        int mutant_index2 = mutant_indices[3*_i + 1];
        int mutant_index3 = mutant_indices[3*_i + 2];
        bool mutate = mutate_indices[i];
        if (mutate) {
            population_new[i] = fabs( 
                population_old[mutant_index1*genome_size + _j] + F*(
                        population_old[mutant_index2*genome_size + _j] -
                        population_old[mutant_index3*genome_size + _j]));
        } else {
            population_new[i] = population_old[i];
        }
    }
}

__global__
void gpu_set_rejection_indices(bool * rejection_indices, double * fitness_new, double * fitness_old, int population_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size) {
        bool accept = fitness_new[i] <= fitness_old[i];
        rejection_indices[i] = accept;
        if (accept) {
            fitness_old[i] = fitness_new[i];
        }
    }
}

__global__
void gpu_swap_control_parameters(double * control_parameter_old, double * control_parameter_new, bool * rejection_indices, int population_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size) {
        if (rejection_indices[i]) {
            control_parameter_old[i] = control_parameter_new[i];
        }
    }
}

__global__
void gpu_swap_populations(double * population_old, double * population_new, bool * rejection_indices, int population_size, int genome_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < population_size*genome_size) {
        int _i = i/genome_size;
        if (rejection_indices[_i]) {
            population_old[i] = population_new[i];
        }
    }
}

// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
// GPU KERNEL WRAPPER --------------------------------------------------------
// ---------------------------------------------------------------------------
// ---------------------------------------------------------------------------
namespace cuda_wrapper {
    #ifdef DEAC_DEBUG
        void gpu_check_array_wrapper(dim3 grid_size, dim3 group_size, double * _array, int length) {
            gpu_check_array <<<grid_size, group_size, 0, 0>>> ( 
                    _array, length
                    );
        }
    
        void gpu_check_array_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * _array, int length) {
            gpu_check_array <<<grid_size, group_size, 0, stream>>> ( 
                    _array, length
                    );
        }
    #endif

    void gpu_matrix_multiply_MxN_by_Nx1_wrapper(dim3 grid_size, dim3 group_size, double * C, double * A, double * B, int N, int idx) {
        gpu_matrix_multiply_MxN_by_Nx1 <<<grid_size, group_size, 0, 0>>> ( 
                C, A, B, N, idx
                );
    }
    void gpu_matrix_multiply_MxN_by_Nx1_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * C, double * A, double * B, int N, int idx) {
        gpu_matrix_multiply_MxN_by_Nx1 <<<grid_size, group_size, 0, stream>>> ( 
                C, A, B, N, idx
                );
    }
    
    void gpu_matrix_multiply_LxM_by_MxN_wrapper(dim3 grid_size, dim3 group_size, double * C, double * A, double * B, int L, int M, int idx) {
        gpu_matrix_multiply_LxM_by_MxN <<<grid_size, group_size, 0, 0>>> ( 
                C, A, B, L, M, idx
                );
    }
    void gpu_matrix_multiply_LxM_by_MxN_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * C, double * A, double * B, int L, int M, int idx) {
        gpu_matrix_multiply_LxM_by_MxN <<<grid_size, group_size, 0, stream>>> ( 
                C, A, B, L, M, idx
                );
    }
    
    void gpu_normalize_population_wrapper(dim3 grid_size, dim3 group_size, double * population, double * normalization, double zeroth_moment, int population_size, int genome_size) {
        gpu_normalize_population <<<grid_size, group_size, 0, 0>>> ( 
                population, normalization, zeroth_moment, population_size, genome_size
                );
    }
    void gpu_normalize_population_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * population, double * normalization, double zeroth_moment, int population_size, int genome_size) {
        gpu_normalize_population <<<grid_size, group_size, 0, stream>>> ( 
                population, normalization, zeroth_moment, population_size, genome_size
                );
    }
    
    void gpu_set_fitness_wrapper(dim3 grid_size, dim3 group_size, double * fitness, double * isf, double * isf_model, double * isf_error, int number_of_timeslices, int idx) {
        gpu_set_fitness <<<grid_size, group_size, 0, 0>>> ( 
                fitness, isf, isf_model, isf_error, number_of_timeslices, idx
                );
    }
    void gpu_set_fitness_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * fitness, double * isf, double * isf_model, double * isf_error, int number_of_timeslices, int idx) {
        gpu_set_fitness <<<grid_size, group_size, 0, stream>>> ( 
                fitness, isf, isf_model, isf_error, number_of_timeslices, idx
                );
    }
    
    void gpu_set_fitness_moments_reduced_chi_squared_wrapper(dim3 grid_size, dim3 group_size, double * fitness, double * moments, double moment, double moment_error, int population_size) {
        gpu_set_fitness_moments_reduced_chi_squared <<<grid_size, group_size, 0, 0>>> ( 
                fitness, moments, moment, moment_error, population_size
                );
    }
    void gpu_set_fitness_moments_reduced_chi_squared_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * fitness, double * moments, double moment, double moment_error, int population_size) {
        gpu_set_fitness_moments_reduced_chi_squared <<<grid_size, group_size, 0, stream>>> ( 
                fitness, moments, moment, moment_error, population_size
                );
    }
    
    void gpu_set_fitness_moments_chi_squared_wrapper(dim3 grid_size, dim3 group_size, double * fitness, double * moments, double moment, int population_size) {
        gpu_set_fitness_moments_chi_squared <<<grid_size, group_size, 0, 0>>> ( 
                fitness, moments, moment, population_size
                );
    }
    void gpu_set_fitness_moments_chi_squared_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * fitness, double * moments, double moment, int population_size) {
        gpu_set_fitness_moments_chi_squared <<<grid_size, group_size, 0, stream>>> ( 
                fitness, moments, moment, population_size
                );
    }
    
    void gpu_get_minimum_fitness_wrapper(dim3 grid_size, dim3 group_size, double * fitness, double * minimum_fitness, int population_size) {
        gpu_get_minimum_fitness <<<grid_size, group_size, 0, 0>>> ( 
                fitness, minimum_fitness, population_size
                );
    }
    void gpu_get_minimum_fitness_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * fitness, double * minimum_fitness, int population_size) {
        gpu_get_minimum_fitness <<<grid_size, group_size, 0, stream>>> ( 
                fitness, minimum_fitness, population_size
                );
    }
    
    void gpu_set_fitness_mean_wrapper(dim3 grid_size, dim3 group_size, double * fitness_mean, double * fitness, int population_size, int idx) {
        gpu_set_fitness_mean <<<grid_size, group_size, 0, 0>>> ( 
                fitness_mean, fitness, population_size, idx
                );
    }
    void gpu_set_fitness_mean_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * fitness_mean, double * fitness, int population_size, int idx) {
        gpu_set_fitness_mean <<<grid_size, group_size, 0, stream>>> ( 
                fitness_mean, fitness, population_size, idx
                );
    }
    
    void gpu_set_fitness_standard_deviation_wrapper(dim3 grid_size, dim3 group_size, double * fitness_standard_deviation, double * fitness_mean, double * fitness, int population_size, int idx) {
        gpu_set_fitness_standard_deviation <<<grid_size, group_size, 0, 0>>> ( 
                fitness_standard_deviation, fitness_mean, fitness, population_size, idx
                );
    }
    void gpu_set_fitness_standard_deviation_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * fitness_standard_deviation, double * fitness_mean, double * fitness, int population_size, int idx) {
        gpu_set_fitness_standard_deviation <<<grid_size, group_size, 0, stream>>> ( 
                fitness_standard_deviation, fitness_mean, fitness, population_size, idx
                );
    }
    
    void gpu_set_fitness_standard_deviation_sqrt_wrapper(dim3 grid_size, dim3 group_size, double * fitness_standard_deviation, int max_generations) {
        gpu_set_fitness_standard_deviation_sqrt <<<grid_size, group_size, 0, 0>>> ( 
                fitness_standard_deviation, max_generations
                );
    }
    void gpu_set_fitness_standard_deviation_sqrt_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * fitness_standard_deviation, int max_generations) {
        gpu_set_fitness_standard_deviation_sqrt <<<grid_size, group_size, 0, stream>>> ( 
                fitness_standard_deviation, max_generations
                );
    }
    
    void gpu_set_population_new_wrapper(dim3 grid_size, dim3 group_size, double * population_new, double * population_old, int * mutant_indices, double * differential_weights_new, bool * mutate_indices, int population_size, int genome_size) {
        gpu_set_population_new <<<grid_size, group_size, 0, 0>>> ( 
                population_new, population_old, mutant_indices, differential_weights_new, mutate_indices, population_size, genome_size
                );
    }
    void gpu_set_population_new_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * population_new, double * population_old, int * mutant_indices, double * differential_weights_new, bool * mutate_indices, int population_size, int genome_size) {
        gpu_set_population_new <<<grid_size, group_size, 0, stream>>> ( 
                population_new, population_old, mutant_indices, differential_weights_new, mutate_indices, population_size, genome_size
                );
    }
    
    void gpu_set_rejection_indices_wrapper(dim3 grid_size, dim3 group_size, bool * rejection_indices, double * fitness_new, double * fitness_old, int population_size) {
        gpu_set_rejection_indices <<<grid_size, group_size, 0, 0>>> ( 
                rejection_indices, fitness_new, fitness_old, population_size
                );
    }
    void gpu_set_rejection_indices_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, bool * rejection_indices, double * fitness_new, double * fitness_old, int population_size) {
        gpu_set_rejection_indices <<<grid_size, group_size, 0, stream>>> ( 
                rejection_indices, fitness_new, fitness_old, population_size
                );
    }
    
    void gpu_swap_control_parameters_wrapper(dim3 grid_size, dim3 group_size, double * control_parameter_old, double * control_parameter_new, bool * rejection_indices, int population_size) {
        gpu_swap_control_parameters <<<grid_size, group_size, 0, 0>>> ( 
                control_parameter_old, control_parameter_new, rejection_indices, population_size
                );
    }
    void gpu_swap_control_parameters_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * control_parameter_old, double * control_parameter_new, bool * rejection_indices, int population_size) {
        gpu_swap_control_parameters <<<grid_size, group_size, 0, stream>>> ( 
                control_parameter_old, control_parameter_new, rejection_indices, population_size
                );
    }
    
    void gpu_swap_populations_wrapper(dim3 grid_size, dim3 group_size, double * population_old, double * population_new, bool * rejection_indices, int population_size, int genome_size) {
        gpu_swap_populations <<<grid_size, group_size, 0, 0>>> ( 
                population_old, population_new, rejection_indices, population_size, genome_size
                );
    }
    void gpu_swap_populations_wrapper(dim3 grid_size, dim3 group_size, hipStream_t stream, double * population_old, double * population_new, bool * rejection_indices, int population_size, int genome_size) {
        gpu_swap_populations <<<grid_size, group_size, 0, stream>>> ( 
                population_old, population_new, rejection_indices, population_size, genome_size
                );
    }
}
